#include "hip/hip_runtime.h"
#include <cfloat>
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/ordinal_regression_loss_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void kernel_ordreg_softmax_forward(const int k,
    const Dtype* x, Dtype* y, const Dtype* label, const Dtype* weight, Dtype* loss) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int sample_idx = idx / k;
  const int label_idx = idx % k;
  const int offset = 2*(sample_idx*k + label_idx);
  const int this_label = static_cast<int>(label[sample_idx]);
  const Dtype this_weight = weight[label_idx];
  const Dtype* x_data = x + offset;
  Dtype* y_data = y + offset;
  Dtype* loss_data = loss + offset;
  Dtype max_input = max(x_data[0], x_data[1]);
  y_data[0] = exp(x_data[0] - max_input);
  y_data[1] = exp(x_data[1] - max_input);
  Dtype sum = y_data[0] + y_data[1];
  y_data[0] /= sum;
  y_data[1] /= sum;
  if (label_idx < this_label) {
    loss_data[0] = 0;
    loss_data[1] = -log(max(y[1], Dtype(FLT_MIN)));
  }
  else {
    loss_data[0] = -log(max(y[0], Dtype(FLT_MIN)));
    loss_data[1] = 0;
  }
  loss_data[0] *= this_weight;
  loss_data[1] *= this_weight;
}

template<typename Dtype>
void OrdinalRegressionLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  int n = bottom[0]->shape(0);
  int nthread = n * k_;
  const Dtype* x = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const Dtype* weight = weight_.gpu_data();
  Dtype* y = prob_.mutable_gpu_data();
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();  // reuse
  kernel_ordreg_softmax_forward<Dtype><<<CAFFE_GET_BLOCKS(nthread),
      CAFFE_CUDA_NUM_THREADS>>>(k_, x, y, label, weight, loss_data);
  Dtype loss;
  caffe_gpu_asum(bottom[0]->count(), loss_data, &loss);
  top[0]->mutable_cpu_data()[0] = loss / n;
}

template<typename Dtype>
void OrdinalRegressionLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(OrdinalRegressionLossLayer);

}  // namespace caffe
